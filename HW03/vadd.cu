#include<iostream>
#include <hip/hip_runtime.h>

__global__ void vadd(const float* a, float* b, unsigned int n) {
	
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		b[i] = a[i] + b[i];
	}
}
