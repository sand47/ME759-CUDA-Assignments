#include<iostream>
#include <hip/hip_runtime.h>

#define N 16

__global__ void sumThredBlk(int* dA) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	dA[index] = threadIdx.x + blockIdx.x;
}

int main(void) {

	int* hA;  // host copies of a 
	int* dA; // device copies of a 
	int size = N * sizeof(int);

	// Alloc space for device copies for dA
	hipMalloc((void**)& dA, size);

	// Alloc space for host copies of a
	hA = (int*)malloc(size); 

	// Copy inputs to device
	hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);

	// Launch sumThredBlk() kernel on GPU with 2 blocks and 8 threads
	sumThredBlk<<<2,8>>>(dA);
	hipDeviceSynchronize();

	// Copy result back to host
	hipMemcpy(hA, dA, size, hipMemcpyDeviceToHost);

	// prints the dA array
	
	for (int i = 0; i < N; i++)
	{
		std::printf("%d ", hA[i]);

	}
	
	// Cleanup
	free(hA);
	hipFree(dA);
	return 0;
}