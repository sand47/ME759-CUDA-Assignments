#include<iostream>
#include <hip/hip_runtime.h>

__global__ void hello_world() {
	std::printf("Hello World! I am thread %d.\n", threadIdx.x);
}

int main() {
	
	hello_world<<<1,4>>>();
	hipDeviceSynchronize();
	return 0;
}