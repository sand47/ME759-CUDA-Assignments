#include<iostream>
#include <hip/hip_runtime.h>
#include "vadd.cuh"


void random_ints(float* a, int m)
{
	int i;
	for (i = 0; i < m; ++i)
		a[i] = 1.0;
}

int main(int argc, char** argv) {

	hipEvent_t startEvent, stopEvent;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);

	const int n = atoi(argv[1]);

	float* a; // host copies of a 
	float* b; // host copies of b
	float* dA; // device copies of a 
	float* dB; // device copies of B

	float size = n * sizeof(float);
	int blocksize = 1;
	
	a = (float*)malloc(size); 
	random_ints(a, n);
	b = (float*)malloc(size); 
	random_ints(b, n);

	hipMalloc((void**)& dA, size);
	hipMalloc((void**)& dB, size);
	
	// Copy inputs to device
	
	hipMemcpy(dA, a, size , hipMemcpyHostToDevice);
	hipMemcpy(dB, b, size , hipMemcpyHostToDevice);

	blocksize = (n +1023)/ 1024;
	
	// Launch add() kernel on GPU with N blocks
	hipEventRecord(startEvent, 0);
	
	vadd<<<blocksize,1024>>>(dA,dB,n);

	hipDeviceSynchronize();
	hipEventRecord(stopEvent, 0);

	hipEventSynchronize(stopEvent);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	
	// Copy result back to host
	hipMemcpy(b, dB, size , hipMemcpyDeviceToHost);
		
	// print time in seconds
		
	std::printf("%f \n",elapsedTime/1000);

	// prints the b array first and last index
	std::printf("%f \n",b[0]);
	std::printf("%f \n",b[n - 1]);
		
	// Cleanup

	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);
	free(a);
	free(b);
	hipFree(dA);
	hipFree(dB);
	return 0;
}