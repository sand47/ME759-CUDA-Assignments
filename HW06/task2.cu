#include "scan.cuh"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

using std::cout;
using std::endl;

int main(int argc, char **argv) {
  unsigned int n = atoi(argv[1]);
  unsigned int threads_per_block = 1024; // should be a multiple of warp size
  float *Hin, *Hout;

  // Timing CUDA events
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Allocating space for arrays on host, Initialized to zero
  Hin = new float[n]();
  Hout = new float[n]();

  float a = -5;
  float b = 5;
  // Setting Random values to the host array
  for (unsigned int i = 0; i < n; ++i) {
    // This generates a random floats in range a to b
    Hin[i] = (float)(a + (((float)rand()) / (float)RAND_MAX) * (b - a));

    // Set a known value, alternatively
    // Hin[i] = (float)(i+1);
  }

  // Calling the reduce operation
  hipEventRecord(start);
  scan(Hin, Hout, n, threads_per_block); // Output is expected to be in Hout
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Calculate time in milliseconds
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  // Prints the last element of the output array
  cout << Hout[n - 1] << endl;

  // Time taken by the full scan method in milliseconds
  cout << milliseconds << endl;

  // Cleanup
  hipFree(Hin);
  hipFree(Hout);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
