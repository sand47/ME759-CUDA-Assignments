#include "mmul.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

// Uses a single cuBLAS call to perform the operation C := A B + C
// handle is a handle to an open cuBLAS instance
// A, B, and C are matrices with n rows and n columns stored in column-major
// NOTE: The cuBLAS call should be followed by a call to hipDeviceSynchronize()
// for timing purposes
void mmul(hipblasHandle_t handle, const float *A, const float *B, float *C,
          int n) {

  float alpha = 1.0f;
  float beta = 1.0f;

  // Single CuBlas call
  // C = alpha * op(A) op (B) + beta * C
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, A, n, B, n,
              &beta, C, n);

  // Wait for GPU to finish
  hipDeviceSynchronize();
}
