#include "hip/hip_runtime.h"
#include "scan.cuh"
#include <iostream>

using std::cout;
using std::endl;
using std::printf;

/*
1. The main hillis_steele algorithm that will work on each block independenty
*/
__global__ void hillis_steele(const float *Din, float *Dout, unsigned int n,
                              float *blockSums, unsigned int total_size) {

  extern __shared__ float temp[];
  unsigned int thid = threadIdx.x;
  unsigned int pout = 0, pin = 1;

  unsigned int arrIdx = threadIdx.x + (blockIdx.x * blockDim.x);

  if (arrIdx <= total_size) {

    // Load data from device to shared memory
    temp[thid] = (arrIdx == 0) ? 0 : Din[arrIdx - 1];
    __syncthreads();

    for (int offset = 1; offset < n; offset *= 2) {

      pout = 1 - pout;
      pin = 1 - pout;

      if (thid >= offset) {
        temp[pout * n + thid] =
            temp[pin * n + thid] + temp[pin * n + thid - offset];
      } else {
        temp[pout * n + thid] = temp[pin * n + thid];
      }

      __syncthreads();
    }

    // If this is the last thread in it's block, write it's output
    // (thid == n) has to be checked for when total elements < threads_per_block
    if ((thid == (blockDim.x - 1)) || (thid == total_size - 1))
      blockSums[blockIdx.x] = temp[pout * n + thid];

    // Copy data back to device memory
    Dout[arrIdx] = temp[pout * n + thid];
  }
}

/*
2. Defining an auxilliary kernel ot accumulate block sums
This may be incorporated in the main hillis_steele kernel
*/
__global__ void block_hillis_steele(const float *Din, float *Dout,
                                    unsigned int n) {

  extern __shared__ float temp[];
  unsigned int thid = threadIdx.x;
  unsigned int pout = 0, pin = 1;

  if (thid <= n) {
    temp[thid] = (thid == 0) ? 0 : Din[thid - 1];

    __syncthreads();

    for (int offset = 1; offset < n; offset *= 2) {

      pout = 1 - pout;
      pin = 1 - pout;

      if (thid >= offset) {
        temp[pout * n + thid] =
            temp[pin * n + thid] + temp[pin * n + thid - offset];
      } else {
        temp[pout * n + thid] = temp[pin * n + thid];
      }

      __syncthreads();
    }

    // Copy data back to device memory
    Dout[thid] = temp[pout * n + thid];
  }
}

/*
3 Adds the previous block sum to every element of output
*/
__global__ void block_adder_kernel(float *Dout, const float *blockSums,
                                   unsigned int total_size) {
  unsigned int arrIdx = threadIdx.x + (blockIdx.x * blockDim.x);

  if (arrIdx <= total_size)
    Dout[arrIdx] += blockSums[blockIdx.x];

  __syncthreads();
}

__host__ void scan(const float *in, float *out, unsigned int n,
                   unsigned int threads_per_block) {

  float *Din, *Dout, *blockSums, *blockSums_out;
  unsigned int shMemSize = (2 * threads_per_block) * sizeof(float);
  hipError_t err = hipSuccess;
  unsigned int n_blocks;

  // Allocate space for device: Din
  err = hipMalloc((void **)&Din, n * sizeof(float));
  if (err != hipSuccess)
    cout << "hipMalloc Failed for Input Array! Error: " << err << endl;

  err = hipMalloc((void **)&Dout, n * sizeof(float));
  if (err != hipSuccess)
    cout << "hipMalloc Failed for Output Array! Error: " << err << endl;

  // We need a separate array to store block sums
  n_blocks = (n + (threads_per_block - 1)) / threads_per_block;
  err = hipMalloc((void **)&blockSums, n_blocks * sizeof(float));
  if (err != hipSuccess)
    cout << "hipMalloc Failed for BlockSums Array! Error: " << err << endl;

  hipMemset(blockSums, 0, n_blocks); // Initializing hipMemset

  // Copying the data to device
  hipMemcpy(Din, in, n * sizeof(float), hipMemcpyHostToDevice);

  // Implementation that supports n upto 1024
  hillis_steele<<<n_blocks, threads_per_block, shMemSize>>>(
      Din, Dout, threads_per_block, blockSums, n);

  err = hipMalloc((void **)&blockSums_out, n_blocks * sizeof(float));
  if (err != hipSuccess)
    cout << "hipMalloc Failed for BlockSums_Out Array! Error: " << err << endl;

  // Running hillis steele algo again, this time on block_sums
  // If we keep the batch_size = 1024, we cannot possibly have n_blocks > 1024
  // because Assumption: n <= threads_per_block * threads_per_block
  block_hillis_steele<<<1, threads_per_block, shMemSize>>>(
      blockSums, blockSums_out, n_blocks);

  block_adder_kernel<<<n_blocks, threads_per_block>>>(Dout, blockSums_out, n);

  // Copy results back to the host
  hipMemcpy(out, Dout, n * sizeof(float), hipMemcpyDeviceToHost);

  // Wait for GPU to finish
  hipDeviceSynchronize();

  // Cleanup
  hipFree(Din);
  hipFree(Dout);
  hipFree(blockSums);
  hipFree(blockSums_out);
}
