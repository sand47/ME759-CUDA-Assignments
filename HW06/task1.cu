#include "mmul.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

using std::cout;
using std::endl;

int main(int argc, char **argv) {
  unsigned int n = atoi(argv[1]);
  unsigned int n_tests = atoi(argv[2]);
  float *A, *B, *C;
  unsigned int size = n * n;

  hipblasHandle_t handle;

  // Timing CUDA events
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // 1. Create matrices of size n*n in managed (unified) memory
  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&A, size * sizeof(float));
  hipMallocManaged(&B, size * sizeof(float));
  hipMallocManaged(&C, size * sizeof(float));

  // 2. Initialize the matrices A, B and C
  // Boundaries for random values
  float low = -10;
  float high = 10;
  for (unsigned int i = 0; i < size; ++i) {
    // This generates a random integers in range -5 to 5
    A[i] = low + (((float)rand()) / (float)RAND_MAX) * (high - low);
    B[i] = low + (((float)rand()) / (float)RAND_MAX) * (high - low);
    C[i] = low + (((float)rand()) / (float)RAND_MAX) * (high - low);
  }

  hipblasCreate(&handle);

  // 3. Set math mode
  hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

  // 4. Call mmul function n_tests times
  hipEventRecord(start);
  for (int j = 0; j < n_tests; ++j) {
    mmul(handle, A, B, C, n);
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Calculate total time in milliseconds
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  // 5. Prints the average time taken to run the mmul in milliseconds
  cout << (milliseconds / n_tests) << endl;

  // Cleanup
  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipblasDestroy(handle);

  return 0;
}
