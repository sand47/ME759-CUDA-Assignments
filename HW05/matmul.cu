#include<iostream>
#include <hip/hip_runtime.h>



__global__ void matmul_kernel(const float* A, const float* B, float* C, unsigned int n)
{
    
    extern __shared__ float sm[];

    float *sA = &sm[0];
    float *sB = &sA[blockDim.x*blockDim.x];
   
    int r = blockIdx.y*blockDim.x+ threadIdx.y;
    int c = blockIdx.x*blockDim.x + threadIdx.x;

    float opval = 0.0;
    
    int tx = threadIdx.x; int ty = threadIdx.y; 
    int bx = blockIdx.x; int by = blockIdx.y;

    for (int q = 0; q < (blockDim.x + n - 1)/blockDim.x; q++)
 {
        	
	if (q*blockDim.x + threadIdx.x < n && r < n)
	 sA[blockDim.x*ty+ tx] = A[r*n + q*blockDim.x + threadIdx.x];
       else
	 sA[blockDim.x*ty+ tx] = 0.0;
	
	if (q*blockDim.x+ threadIdx.y < n && c< n)  
	 sB[blockDim.x*ty+ tx] = B[(q*blockDim.x + threadIdx.y)*n + c];
        else 
	 sB[blockDim.x*ty+ tx] = 0.0;

        __syncthreads();

        for (int j = 0; j < blockDim.x; ++j)//Multiplying Elements present in tile
        {
            opval += sA[blockDim.x*ty+ j] * sB[blockDim.x*j+ tx];
        }

        __syncthreads();
    }
    
    
    int cindex = (by * blockDim.x + threadIdx.y)*n+(bx*blockDim.x)+threadIdx.x;
    if (r < n && c < n)
	C[cindex]=opval;
    

}

__host__ void matmul(const float* A, const float* B, float* C, unsigned int n, unsigned int block_dim)

{
    dim3 dimBlock(block_dim,block_dim);
    int gridsize= (n + block_dim -1)/ block_dim;
    dim3 dimGrid(gridsize,gridsize);
   
    matmul_kernel<<<dimGrid,dimBlock, 2*block_dim*block_dim* sizeof(float)>>>(A, B, C, n);
    hipDeviceSynchronize();

}



