#include<iostream>
#include <hip/hip_runtime.h>


__global__ void reduce_kernel(const int* g_idata, int* g_odata, unsigned int n)

{
    extern __shared__ int sdata[];

    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
   if(i<n)
     {

    sdata[threadIdx.x] = g_idata[i];
     }
    __syncthreads();
    
   for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
    if (threadIdx.x< s) {
	if(i+s<n)
	{
        sdata[threadIdx.x] += sdata[threadIdx.x+ s];
	}
    }
    __syncthreads();
   }

    
   
    if(threadIdx.x==0)
     {
	g_odata[blockIdx.x] = sdata[0];	
     }
	 	 
}


__host__ int reduce(const int* arr, unsigned int N, unsigned int threads_per_block)
{
	
	int gdim = (N +threads_per_block-1)/ (threads_per_block);
	
	int* dev_i;
	int* dev_o;
	
	int op =0;
		
	hipMallocManaged(&dev_i,N * sizeof(int));
	hipMallocManaged(&dev_o,gdim * sizeof(int));
	
	hipMemcpy(dev_i, arr, N * sizeof(int), hipMemcpyHostToDevice);
	
	int size = N;
	int g = gdim;
	
	for(int i=0;i<g;i++)
	{
	reduce_kernel<<<gdim,threads_per_block,threads_per_block*sizeof(int)>>>(dev_i,dev_o,size);
	
        hipDeviceSynchronize();
	size = gdim;
	
	int *temp = dev_o;
	dev_o = dev_i;
	dev_i = temp;	
	
	gdim = (size +threads_per_block-1)/ (threads_per_block);
    	
 		
	}
	
	op = dev_i[0];	
	
	hipFree(dev_i);
        hipFree(dev_o);
	
	return op;
}