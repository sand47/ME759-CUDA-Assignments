#include<iostream>
#include <hip/hip_runtime.h>
#include "reduce.cuh"

int main(int argc, char** argv) {

	hipEvent_t startEvent, stopEvent;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
	
	unsigned int n = atoi(argv[1]);
	unsigned int thread_per_block = atoi(argv[2]);
	
	int* A = new int[n];
	int sum;

	for (unsigned int i = 0; i < n; i++)
	{
		A[i] = 1;
	}

	hipEventRecord(startEvent, 0);
	
	sum = reduce(A,n,thread_per_block);
	
	hipEventRecord(stopEvent, 0);

	hipEventSynchronize(stopEvent);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);

	std::printf("%d \n", sum);
	std::printf("%f \n", elapsedTime);

	// Cleanup

	delete[] A;
	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);
	return 0;

}