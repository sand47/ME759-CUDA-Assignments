#include <iostream>

#define HIPCUB_STDERR // print CUDA runtime errors to console
#include <cub/device/device_reduce.cuh>
#include <cub/util_allocator.cuh>
#include <stdio.h>
// #include "test/test_util.h"
using namespace hipcub;
CachingDeviceAllocator g_allocator(true); // Caching allocator for device memory

using std::cout;
using std::endl;
using namespace hipcub;

int main(int argc, char **argv) {

  unsigned int num_items = atoi(argv[1]);
  bool check_sum = false;

  // Set up host arrays
  int *h_in = new int[num_items]();

  // Timing CUDA events
  hipEvent_t start, stop;
  float milliseconds = 0;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Filling with random values between low and high
  int low = -100, high = 100;
  // Setting Random values to the host array
  for (unsigned int i = 0; i < num_items; ++i) {
    // This generates a random integers in range low to high
    h_in[i] = (int)(low + (((float)rand()) / (float)RAND_MAX) * (high - low));
  }

  int sum = 0;
  // Sum as calculated on the CPU
  if (check_sum) {
    for (unsigned int i = 0; i < num_items; i++)
      sum += h_in[i];
    cout << "cpu_sum is: " << sum << endl;
  }

  // Set up device arrays
  int *d_in = NULL;
  CubDebugExit(
      g_allocator.DeviceAllocate((void **)&d_in, sizeof(int) * num_items));

  // Initialize device input
  CubDebugExit(
      hipMemcpy(d_in, h_in, sizeof(int) * num_items, hipMemcpyHostToDevice));

  // Setup device output array
  int *d_sum = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void **)&d_sum, sizeof(int) * 1));

  // Request and allocate temporary storage
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in,
                                 d_sum, num_items));
  CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

  // Do the actual reduce operation -- measure time using CUDA events
  hipEventRecord(start);
  CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in,
                                 d_sum, num_items));
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  int gpu_sum;
  CubDebugExit(
      hipMemcpy(&gpu_sum, d_sum, sizeof(int) * 1, hipMemcpyDeviceToHost));
  // Check for correctness
  if (check_sum) {
    printf("%s\n", (gpu_sum == sum ? "Test passed." : "Test failed."));
  }

  hipEventElapsedTime(&milliseconds, start, stop);
  // 4. Prints the resulting sum
  cout << gpu_sum << endl;

  // 5. Prints the time taken to run the reduction in milliseconds
  cout << milliseconds << endl;

  // Cleanup
  if (d_in)
    CubDebugExit(g_allocator.DeviceFree(d_in));
  if (d_sum)
    CubDebugExit(g_allocator.DeviceFree(d_sum));
  if (d_temp_storage)
    CubDebugExit(g_allocator.DeviceFree(d_temp_storage));

  return 0;
}
