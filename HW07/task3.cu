#include <iostream>

// Thrust headers
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "count.cuh"

using std::cout;
using std::endl;

int main(int argc, char **argv) {
  unsigned int n = atoi(argv[1]);

  // Timing CUDA events
  hipEvent_t start, stop;
  float milliseconds = 0;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Allocating a host vector of size n
  thrust::host_vector<int> h_in((size_t)n);

  // Filling with random values between low and high
  int low = 0, high = 1000;
  // Setting Random values to the host array
  for (unsigned int i = 0; i < n; ++i) {
    // This generates a random floats in range low to high
    h_in[i] = (int)(low + (((float)rand()) / (float)RAND_MAX) * (high - low));
  }

  // Copy from host to device
  thrust::device_vector<int> d_in = h_in;
  thrust::device_vector<int> values(n);
  thrust::device_vector<int> counts(n);

  hipEventRecord(start);
  count(d_in, values, counts);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Print the last element of the values array
  cout << *(values.end() - 1) << endl;

  // Print the last element of the counts array
  cout << *(counts.end() - 1) << endl;

  // Prints the time taken to run the scan in milliseconds
  hipEventElapsedTime(&milliseconds, start, stop);
  cout << milliseconds << endl;

  return 0;
}
