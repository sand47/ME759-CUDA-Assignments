#include <iostream>

#define HIPCUB_STDERR // print CUDA runtime errors to console
#include <cub/device/device_scan.cuh>
#include <cub/util_allocator.cuh>
#include <stdio.h>

using std::cout;
using std::endl;
using namespace hipcub;
CachingDeviceAllocator g_allocator(true); // Caching allocator for device memory

int main(int argc, char **argv) {

  unsigned int n = atoi(argv[1]);

  // Timing CUDA events
  hipEvent_t start, stop;
  float milliseconds = 0;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Set up host arrays
  float *h_in = new float[n]();
  float *h_out = new float[n]();

  // Filling with random values between low and high
  float low = -100, high = 100;
  // Setting Random values to the host array
  for (unsigned int i = 0; i < n; ++i) {
    // This generates a random floats in range -5 to 5
    h_in[i] = (float)(low + (((float)rand()) / (float)RAND_MAX) * (high - low));
  }

  // 1. Set up device input/output array
  float *d_in = NULL, *d_out = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void **)&d_in, sizeof(float) * n));
  // Copy data from the host array into device
  CubDebugExit(
      hipMemcpy(d_in, h_in, sizeof(float) * n, hipMemcpyHostToDevice));

  // Set up device output array
  CubDebugExit(g_allocator.DeviceAllocate((void **)&d_out, sizeof(float) * n));
  CubDebugExit(hipMemset(d_out, 0, sizeof(float) * n));

  // Request and allocate temporary storage
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  CubDebugExit(DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
                                        d_in, d_out, n));
  CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

  // Do the actual scan operation -- measure time using CUDA events
  hipEventRecord(start);
  CubDebugExit(DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
                                        d_in, d_out, n));
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Copy data back to the host
  CubDebugExit(
      hipMemcpy(h_out, d_out, sizeof(float) * n, hipMemcpyDeviceToHost));

  hipEventElapsedTime(&milliseconds, start, stop);
  // 4. Prints the last element of the output array
  cout << h_out[n - 1] << endl;

  // 5. Prints the time taken to run the scan in milliseconds
  cout << milliseconds << endl;

  return 0;
}
