#include <iostream>

// Thrust headers
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

using std::cout;
using std::endl;

int main(int argc, char **argv) {
  unsigned int n = atoi(argv[1]);

  // Timing CUDA events
  hipEvent_t start, stop;
  float milliseconds = 0;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // 1. Allocating a host vector of size n
  thrust::host_vector<float> H(n);

  // Filling with random values between low and high
  float low = -100, high = 100;
  // Setting Random values to the host array
  for (unsigned int i = 0; i < n; ++i) {
    // This generates a random floats in range low to high
    H[i] = (float)(low + (((float)rand()) / (float)RAND_MAX) * (high - low));
  }

  // 2. Copy from host to device
  thrust::device_vector<float> D = H;

  // Allocating an output device vector of size n
  thrust::device_vector<float> Dout(n);

  // 3. Call the thrust:exclusive_scan function
  hipEventRecord(start);
  thrust::exclusive_scan(D.begin(), D.end(), Dout.begin());
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // 4. Prints the last element
  cout << Dout[n - 1] << endl;

  // 5. Prints the time taken to run the scan in milliseconds
  hipEventElapsedTime(&milliseconds, start, stop);
  cout << milliseconds << endl;

  return 0;
}
