#include <iostream>

// Thrust headers
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

using std::cout;
using std::endl;

int main(int argc, char **argv) {
  // ./task1 n
  unsigned int n = atoi(argv[1]);

  // Timing CUDA events
  hipEvent_t start, stop;
  float milliseconds = 0;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  // 1. Allocating a host vector of size n
  thrust::host_vector<int> H(n);

  // Filling with random values between low and high
  int low = -100, high = 100;
  // Setting Random values to the host array
  for (unsigned int i = 0; i < n; ++i) {
    // This generates a random integers in range -100 to 100
    H[i] = (int)(low + (((float)rand()) / (float)RAND_MAX) * (high - low));
  }

  // 2. Copy from host to device
  thrust::device_vector<int> D = H;

  // 3. Call thrust::reduce function to do reduction
  hipEventRecord(start);
  int sum = thrust::reduce(D.begin(), D.end(), (int)0, thrust::plus<int>());
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&milliseconds, start, stop);

  // 4. Prints the resulting sum
  cout << sum << endl;

  // 5. Prints the time taken to run the reduction in milliseconds
  cout << milliseconds << endl;

  return 0;
}
