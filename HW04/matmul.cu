#include<iostream>
#include <hip/hip_runtime.h>


__global__ void matmul_kernel(const float* A, const float* B, float* C, size_t n)
{
	
	size_t col = blockIdx.x * blockDim.x + threadIdx.x;
	size_t row =0;
	if (col < n*n) {
		
		row = col / n;
		col = col % n;
	
		for (size_t i = 0; i < n; i++) {
			C[row * n + col] += A[row * n + i] * B[i * n + col];
		}
	}

}

void matmul(const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block)

{

	size_t blocksize = (n*n + threads_per_block - 1) / threads_per_block;
	matmul_kernel <<<blocksize, threads_per_block >>> (A, B, C, n);
	hipDeviceSynchronize();

}



