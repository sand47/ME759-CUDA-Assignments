#include<iostream>
#include <hip/hip_runtime.h>
#include <random>
#include "stencil.cuh"


int main(int argc, char** argv) {

	hipEvent_t startEvent, stopEvent;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);

	srand( (unsigned)time( NULL ) );

	unsigned int n = atoi(argv[1]);
	unsigned int R = atoi(argv[2]); 
	unsigned int thread = atoi(argv[3]);

	float* image;
	float* mask;
	float* output;

	hipMallocManaged(& image, n * sizeof(float));
	hipMallocManaged(& mask, (2*R+1) * sizeof(float));
	hipMallocManaged(& output, n * sizeof(float));

	
	for (unsigned int i = 0; i <n; i++)
	{
		image[i] =(float) rand()/RAND_MAX;		
	}
	
	for(unsigned int j = 0;j <(2*R+1);j++)
	{
		mask[j] = (float) rand()/RAND_MAX;
	}

	
	hipEventRecord(startEvent, 0);
	
	stencil(image, mask, output, n, R,thread);
	
	hipEventRecord(stopEvent, 0);

	hipEventSynchronize(stopEvent);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	
	// print results 
	
	std::printf("%f \n", output[n -1]);
	std::printf("%f \n", elapsedTime);
	
	// Cleanup

	hipFree(image);
	hipFree(mask);
	hipFree(output);
	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);
	return 0;
}