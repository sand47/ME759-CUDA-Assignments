#include<iostream>
#include <hip/hip_runtime.h>
#include "matmul.cuh"


int main(int argc, char** argv) {

	hipEvent_t startEvent, stopEvent;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);

	unsigned int n = atoi(argv[1]);
	unsigned int thread = atoi(argv[2]);
	

	float* A;
	float* B;
	float* C;

	
	hipMallocManaged(& A, n * n * sizeof(float));
	hipMallocManaged(& B, n * n * sizeof(float));
	hipMallocManaged(& C, n * n * sizeof(float));

	for (unsigned int i = 0; i < n * n; i++)
	{
		A[i] = 0.5;
		B[i] = 0.5;
	}

	hipEventRecord(startEvent, 0);
	
	matmul(A, B, C, n, thread);
	
	hipEventRecord(stopEvent, 0);

	hipEventSynchronize(stopEvent);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);

	std::printf("%f \n", C[n * n - 1]);
	std::printf("%f \n", elapsedTime);

	// Cleanup

	hipFree(A);
	hipFree(B);
	hipFree(C);
	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);
	return 0;
}