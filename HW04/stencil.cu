#include<iostream>
#include <hip/hip_runtime.h>


__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R)

{	
       
   extern __shared__ float shared[];

   float opsum=0;
   int flag=(int)R;
   float* mk = &shared[0];
   float* ip = &mk[2*R+1];
   float* op = &ip[blockDim.x+2*R + 1];

   int id =  blockIdx.x * blockDim.x + threadIdx.x;

    if (id <n)
   {
     ip[threadIdx.x+R] = image[id];
   }
   else
   {
     ip[threadIdx.x + R]= 0;
   }
    
   if (threadIdx.x < 2*R+1)
   {
     mk[threadIdx.x] = mask[threadIdx.x];
   }

    if (threadIdx.x < R)
    {
      
      if (id -flag>0)
      {
 	ip[threadIdx.x] = image[id -flag];

      }
      else
      {
    	ip[threadIdx.x]=0;
      }
    

     if ( id + blockDim.x < n)
      {
      	ip[blockDim.x+ threadIdx.x+R] = image[blockDim.x+ id ];
      }
      else
      {
     	 ip[blockDim.x+ threadIdx.x + R] = 0;
      }

    	
    }
    
    __syncthreads();

    for (int k = 0; k < (2*R+1); k++)
    {
        opsum += ip[threadIdx.x+k] * mk[k];
    }
    
    op[threadIdx.x] = opsum;

    if (id<n)
       output[id] = op[threadIdx.x];
   
}

__host__ void stencil(const float* image,const float* mask,float* output,unsigned int n,unsigned int R,unsigned int threads_per_block)
{
   
	int blocksize = (n + threads_per_block - 1) / threads_per_block;
	int size = 2*threads_per_block*sizeof(float)+2*R*sizeof(float) + (2*R+1)*sizeof(float);
	stencil_kernel <<<blocksize, threads_per_block,size>>>(image, mask, output, n, R);
	hipDeviceSynchronize();

}



